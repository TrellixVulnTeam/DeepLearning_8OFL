// -*- c++ -*-
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <ctime>

//http://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf is wrong

__global__ void add(int *a, int*b, int count){
  int id = blockIdx.x + blockDim.x + threadIdx.x;
  if(id < count){
    a[id] += b[id];
  }
}

using namespace std;

int main(int argc, char** argv){
  srand(time(NULL));
  int count = 100;
  int *h_a = new int[count];
  int *h_b = new int[count];

  for(int i=0;i<count;i++){
    h_a[i] = rand() % 1000;
    h_b[i] = rand() % 1000;
  }
  
  int *d_a, *d_b;
  
  hipMalloc(&d_a, sizeof(int)*count);
  hipMalloc(&d_b, sizeof(int)*count);


  hipMemcpy(d_a, h_a, sizeof(int)*count,hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, sizeof(int)*count,hipMemcpyHostToDevice);

  add<<<count/256+1,256>>>(d_a, d_b,count);
  
  hipMemcpy(h_a, d_a, sizeof(int)*count, hipMemcpyDeviceToHost);

  for(int i=0;i<count;i++){
    cout<<h_a[i]<<","<<endl;
  }
  hipFree(d_a);
  hipFree(d_b);
  
  delete[] h_a;
  delete[] h_b;


  return 0;
}

