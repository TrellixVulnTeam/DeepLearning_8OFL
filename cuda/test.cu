#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream.h>
#include <stdlib.h>
#include <ctime>

//http://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf is wrong

__global__ void add(int *a, int*b, int*c){
  *c = *b + *a;
}

int main(int argc, char** argv){
  int a,b,c;
  int *d_a, *d_b, *d_c;

  hipMalloc((void **)&d_a, sizeof(void**));
  hipMalloc((void **)&d_b, sizeof(void**));
  hipMalloc((void **)&d_c, sizeof(void**));
  a = 2;
  b = 7;

  hipMemcpy(d_a, &a, sizeof(d_a),hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, sizeof(d_b),hipMemcpyHostToDevice);

  add<<<1,1>>>(d_a, d_b, d_c);
  hipMemcpy(&c, d_c, sizeof(d_c), hipMemcpyDeviceToHost);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  printf("%d", c);
  printf("asdfasdfasdf\n");
  return 0;
}

